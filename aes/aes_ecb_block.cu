#include "hip/hip_runtime.h"
#include "aes_ecb_block.h"
#include "common.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdint.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>


namespace aes {

	namespace block_level {

		using aes::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		// S table
		static const uint8_t sbox[256] = {
			0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5,
			0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
			0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0,
			0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
			0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc,
			0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
			0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a,
			0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
			0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0,
			0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
			0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b,
			0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
			0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85,
			0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
			0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5,
			0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
			0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17,
			0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
			0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88,
			0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
			0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c,
			0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
			0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9,
			0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
			0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6,
			0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
			0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e,
			0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
			0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94,
			0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
			0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68,
			0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
		};

		// inv S table
		static const uint8_t sboxinv[256] = {
			0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38,
			0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
			0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87,
			0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
			0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d,
			0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
			0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2,
			0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
			0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16,
			0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
			0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda,
			0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
			0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a,
			0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
			0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02,
			0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
			0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea,
			0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
			0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85,
			0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
			0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89,
			0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
			0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20,
			0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
			0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31,
			0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
			0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d,
			0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
			0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0,
			0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
			0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26,
			0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d
		};


		// x-time operation
		__device__ uint8_t rj_xtime(uint8_t x) {
			return ((x << 1) ^ (((x >> 7) & 1) * 0x1b));
		}


		// subbyte operation
		__device__ void aes_subBytes(uint8_t *buf,uint8_t* s_sbox) {
			uint8_t i, b;
			for (i = 0; i < 16; ++i) {
				b = buf[i];
				buf[i] = s_sbox[b];
			}
		}


		// inv subbyte operation
		__device__ void aes_subBytes_inv(uint8_t *buf, uint8_t* s_sbox) {
			uint8_t i, b;
			for (i = 0; i < 16; ++i) {
				b = buf[i];
				buf[i] = s_sbox[b];
			}
		}


		// add round key operation
		__device__ void aes_addRoundKey(uint8_t *buf, uint8_t *key) {
			uint8_t i = 16;
			while (i--) {
				buf[i] ^= key[i];
			}
		}


		// shift row operation
		//  typically thought of in a 2 matrix fashion
		//  memory laid out below
		//    HAVE         WANT
		//  0 4 8  12   0  4  8  12
		//  1 5 9  13   5  9  13 1
		//  2 6 10 14   10 14 2  6
		//  3 7 11 15   15 3  7  11
 		//
		__device__ void aes_shiftRows(uint8_t *buf) {
			uint8_t i, j;
			i = buf[1];
			buf[1] = buf[5];
			buf[5] = buf[9];
			buf[9] = buf[13];
			buf[13] = i;
			i = buf[10];
			buf[10] = buf[2];
			buf[2] = i;
			j = buf[3];
			buf[3] = buf[15];
			buf[15] = buf[11];
			buf[11] = buf[7];
			buf[7] = j;
			j = buf[14];
			buf[14] = buf[6];
			buf[6] = j;
		}

		// inv shift row operation
		__device__ void aes_shiftRows_inv(uint8_t *buf)
		{
			uint8_t i, j;
			i = buf[1];
			buf[1] = buf[13];
			buf[13] = buf[9];
			buf[9] = buf[5];
			buf[5] = i;
			i = buf[2];
			buf[2] = buf[10];
			buf[10] = i;
			j = buf[3];
			buf[3] = buf[7];
			buf[7] = buf[11];
			buf[11] = buf[15];
			buf[15] = j;
			j = buf[6];
			buf[6] = buf[14];
			buf[14] = j;
		}

		// mix column operation
		__device__ void aes_mixColumns(uint8_t *buf) {
			uint8_t i, a, b, c, d, e;
			for (i = 0; i < 16; i += 4)
			{
				a = buf[i];
				b = buf[i + 1];
				c = buf[i + 2];
				d = buf[i + 3];
				e = a ^ b ^ c ^ d;
				buf[i] ^= e ^ rj_xtime(a^b);
				buf[i + 1] ^= e ^ rj_xtime(b^c);
				buf[i + 2] ^= e ^ rj_xtime(c^d);
				buf[i + 3] ^= e ^ rj_xtime(d^a);
			}
		}


		// inv mix column operation
		__device__ void aes_mixColumns_inv(uint8_t *buf) {
			uint8_t i, a, b, c, d, e, x, y, z;
			for (i = 0; i < 16; i += 4)
			{
				a = buf[i];
				b = buf[i + 1];
				c = buf[i + 2];
				d = buf[i + 3];
				e = a ^ b ^ c ^ d;
				z = rj_xtime(e);
				x = e ^ rj_xtime(rj_xtime(z^a^c));
				y = e ^ rj_xtime(rj_xtime(z^b^d));
				buf[i] ^= x ^ rj_xtime(a^b);
				buf[i + 1] ^= y ^ rj_xtime(b^c);
				buf[i + 2] ^= x ^ rj_xtime(c^d);
				buf[i + 3] ^= y ^ rj_xtime(d^a);
			}
		}

		// aes encrypt algorithm one thread/one block with AES_BLOCK_SIZE 
		__global__ void kern_aes_encrypt_ecb(uint8_t *buf_d, uint8_t *key_d,uint8_t* sbox_d, uint32_t numbytes, uint8_t rounds,uint8_t key_length) {
			
			uint8_t buf_t[AES_BLOCK_SIZE]; // stores the blocks data to operate on

			__shared__ uint8_t key_s[240]; // stores the unique key
			__shared__ uint8_t s_sbox[256];

			uint32_t offset = (blockIdx.x * blockDim.x) + threadIdx.x;
			uint32_t buff_offset = offset * AES_BLOCK_SIZE;
			if (buff_offset >= numbytes) { return; }

			if ((offset % blockSize1d) == 0 ) 
			{
				memcpy(key_s, key_d, sizeof(uint8_t)*key_length);
				memcpy(s_sbox, sbox_d, sizeof(uint8_t) * 256);
			}

			memcpy(buf_t, &buf_d[buff_offset], AES_BLOCK_SIZE);

			__syncthreads();

			aes_addRoundKey(buf_t, &key_s[0]);

			for (uint8_t i = 1; i < rounds; ++i)
			{
				aes_subBytes(buf_t,s_sbox);
				aes_shiftRows(buf_t);
				aes_mixColumns(buf_t);
				aes_addRoundKey(buf_t, &key_s[i * AES_BLOCK_SIZE]);
			}
			aes_subBytes(buf_t,s_sbox);
			aes_shiftRows(buf_t);
			aes_addRoundKey(buf_t, &key_s[key_length- AES_BLOCK_SIZE]);

			/* copy thread buffer back into global memory */
			memcpy(&buf_d[buff_offset], buf_t, AES_BLOCK_SIZE);
			__syncthreads();
		}

		// aes decrypt algorithm
		__global__ void kern_aes_decrypt_ecb(uint8_t *buf_d, uint8_t *key_d, uint8_t* sbox_d, uint32_t numbytes, uint8_t rounds, uint8_t key_length) {
			
			uint8_t buf_t[AES_BLOCK_SIZE]; // stores the blocks data to operate on

			__shared__ uint8_t key_s[240]; // stores the unique key
			__shared__ uint8_t s_sbox[256];

			uint32_t offset = (blockIdx.x * blockDim.x) + threadIdx.x;
			uint32_t buff_offset = offset * AES_BLOCK_SIZE;
			if (buff_offset >= numbytes) { return; }

			if ((offset % blockSize1d) == 0)
			{
				memcpy(key_s, key_d, sizeof(uint8_t)*key_length);
				memcpy(s_sbox, sbox_d, sizeof(uint8_t) * 256);
			}

			memcpy(buf_t, &buf_d[buff_offset], AES_BLOCK_SIZE);

			__syncthreads();


			aes_addRoundKey(buf_t, &key_s[key_length-AES_BLOCK_SIZE]);

			for (int round = (rounds - 1); round > 0; --round)
			{
				aes_shiftRows_inv(buf_t);
				aes_subBytes_inv(buf_t,s_sbox);
				aes_addRoundKey(buf_t, &key_s[round * AES_BLOCK_SIZE]);
				aes_mixColumns_inv(buf_t);
			}

			aes_shiftRows_inv(buf_t);
			aes_subBytes_inv(buf_t,s_sbox);
			aes_addRoundKey(buf_t, &key_s[0]);

			/* copy thread back into global memory */
			memcpy(&buf_d[buff_offset], buf_t, AES_BLOCK_SIZE);
			__syncthreads();
		}

		// block level parallelization. Every thread gets one 128 bit block
		int aes::block_level::aes_encrypt_block(aes_info* aes)
		{
			uint8_t* buf_d;
			uint8_t* key_d;
			uint8_t* sbox_d;
			//printf("\nBeginning block level parralelization encryption...\n");

			// get our space
			hipMalloc((void**)&buf_d, sizeof(uint8_t) * aes->padded_length);
			hipMalloc((void**)&key_d, sizeof(uint8_t) * aes->expand_length);
			hipMalloc((void**)&sbox_d, sizeof(uint8_t) * 256);
			checkCUDAError("hipMalloc");
			
			// copy data to device
			hipMemcpy(key_d, aes->key_expand, sizeof(uint8_t) * aes->expand_length, hipMemcpyHostToDevice);
			hipMemcpy(buf_d, aes->data, sizeof(uint8_t) * aes->padded_length, hipMemcpyHostToDevice);
			hipMemcpy(sbox_d, sbox, sizeof(uint8_t) * 256, hipMemcpyHostToDevice);
			checkCUDAError("hipMalloc");

			// calculate how many threads we need to have to have one thread per block
			const int active_threads = aes->padded_length / AES_BLOCK_SIZE;
			dim3 dimBlock = (active_threads + blockSize1d - 1) / blockSize1d;

			//start timer
			timer().startGpuTimer();

			// decryption kernel
			kern_aes_encrypt_ecb << <dimBlock, blockSize1d >> > (buf_d, key_d,sbox_d, aes->padded_length, aes->rounds,aes->expand_length );

			//end timer
			timer().endGpuTimer();

			// copy back results
			hipMemcpy(aes->data, buf_d, sizeof(uint8_t) * aes->padded_length, hipMemcpyDeviceToHost);

			// clean up buffers
			hipFree(buf_d);
			hipFree(key_d);
			hipFree(sbox_d);

			return EXIT_SUCCESS;
		}

		// byte level parallelization. Every thread gets one byte into the 128 bit block
		int aes::block_level::aes_decrypt_block(aes_info* aes)
		{
			uint8_t* buf_d;
			uint8_t* key_d;
			uint8_t* sbox_d;
			//printf("\nBeginning block level parralelization decryption...\n");

			// get our space
			hipMalloc((void**)&buf_d, sizeof(uint8_t) * aes->padded_length);
			hipMalloc((void**)&key_d, sizeof(uint8_t) * aes->expand_length);
			hipMalloc((void**)&sbox_d, sizeof(uint8_t) * 256);

			// copy data to device
			hipMemcpy(key_d, aes->key_expand, sizeof(uint8_t) * aes->expand_length, hipMemcpyHostToDevice);
			hipMemcpy(buf_d, aes->data, sizeof(uint8_t) * aes->padded_length, hipMemcpyHostToDevice);
			hipMemcpy(sbox_d, sboxinv, sizeof(uint8_t) * 256, hipMemcpyHostToDevice);

			// 
			const int active_threads = aes->padded_length / AES_BLOCK_SIZE;
			dim3 dimBlock = (active_threads+ blockSize1d-1) / blockSize1d;
			
			//start timer
			timer().startGpuTimer();

			// decryption kernel
			kern_aes_decrypt_ecb << <dimBlock, blockSize1d >> > (buf_d, key_d,sbox_d, aes->padded_length, aes->rounds, aes->expand_length);

			//end timer
			timer().endGpuTimer();

			// copy back results
			hipMemcpy(aes->data, buf_d, sizeof(uint8_t) * aes->padded_length, hipMemcpyDeviceToHost);

			// clean up buffers
			hipFree(buf_d);
			hipFree(key_d);

			return EXIT_SUCCESS;
		}
	}
}