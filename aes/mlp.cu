#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "mlp.h"
#include <math.h>

#define NUM_LAYERS 2
#define INPUT_NODES 2
#define NUM_WEIGHTS 2

#define blockSize 128

namespace CharacterRecognition {
    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

	void memory_debug_float(int elements, float* cuda_mem, float* cpu_mem)
	{
		printf("elements %d\n ", elements);
		hipMemcpy(cpu_mem, cuda_mem, elements * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("debug failed!");
		printf("=============================\n");
		for (int i = 0; i < elements; i++)
		{
			printf("out[%d] %f \n", i, cpu_mem[i]);
		}
		printf("=============================\n");
	}

        
    // TODO: __global__

	__global__ void kernel_feed_forward(int n, float* dev_in, float* weights)
	{
		int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (tid >= n)
		{
			return;
		}

		float data = dev_in[tid];

		dev_in[tid] = (data * weights[tid]) + (data * weights[tid+n]);
	}

	__global__ void kernel_activate(int n, float* dev_in)
	{
		int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (tid >= n)
		{
			return;
		}

		float var = (dev_in[tid] * -1);
		float e = expf(var);
		dev_in[tid] = 1 / (1 + e);
	}

	float transfer_function(float in)
	{
		float var = (in * -1);
		float e = exp(var);
		return( 1 / (1 + e) );
	}

	//float transfer_derivative(float var)
	//{
	//	float var = (var * -1);
	//	float e = exp(var);
	//	return(1 - (1 / (1 + e)));
	//}

	void feed_forward(float* in, float* out, float* weights, int length)
	{
		for (int i = 0; i < length; i++)
		{
			float temp = in[i];
			out[i] = 0;

			for (int j = 0; j < INPUT_NODES; j++) // or layers?
			{
				float weight = *((weights+i* INPUT_NODES) + j);
				out[i] += (temp * weight);
				printf("in[%d] = %f * %f\n", i, temp, weight);
			}

			printf("feed[%d] = %f\n", i, out[i]);
			out[i] = transfer_function(out[i]);
			printf("activate: %f\n", out[i]);
		}
	}

	//void activate_function(float* in,float* out, int length)
	//{
	//	for (int i = 0; i < length; i++)
	//	{
	//		float var = (in[i] * -1);
	//		float e = exp(var);
	//		out[i] = 1 / (1 + e);
	//		printf("activate: %f\n", in[i]);
	//	}
	//}

	//void update_weights(float* delta_weights, float* weights, float* gradient, float* data, int length)
	//{
	//	for (int i = 0; i < 6; i++)
	//	{

	//	}
	//}

	//float calculate_hidden_gradient( float* weight, float* gradient )
	//{
	//	weight[]
	//	return (delta * transfer_derivative(out));
	//}

	//float calculate_gradient( float out, float target)
	//{
	//	
	//	return (out*target); // I think 1 / e + x 
	//}

	//void back_propagate(float* data, int length,float* weights, float expected_value, float* gradient, float* delta_weight)
	//{
	//	float error = 0;
	//	// loop through and compute the rms not including the bias node 
	//	for (int i = 0; i < length; i++)
	//	{
	//		// get the delta between what we predicted vs expected
	//		float delta = data[i] - expected_value;

	//		error = delta * delta;
	//	}
	//	error /= length;
	//	error = std::sqrt(error);

	//	// calcuate gradient on input layer?
	//	// only have one output layer node thing
	//	for (int n = 4; n < 6; n++)
	//	{
	//		// calculate gradient of the layer
	//		gradient[n] = calculate_gradient( weights[n], error );
	//	}

	//	//calculate gradient on hidden layer?

	//	for (int n = 0; n < 4; n+=2)
	//	{
	//		// calculate hidden layer
	//		calculate_hidden_gradient(weights[n], gradient[6-1-n], data[n]);
	//	}

	//	// update the weights
	//	update_weights();

	//}

    /**
        * Example of use case (follow how you did it in stream compaction)
        */
    void train(int n, int *data, int expected) {
        timer().startGpuTimer();
        
		float weights[6];
		float in_data[4] = { 0,0 };
		float out_data[4];
		float temp[4];
		weights[0] = 10.1;
		weights[1] = .9;
		weights[2] = 20;
		weights[3] = .87;
		weights[4] = 41;
		weights[5] = -54;
		int wt_idx = 0;

		float* dev_buff;
		float* dev_wts;

		float* host_buff = new float[4];

		int rounded_depth = ilog2ceil(NUM_LAYERS);
		int rounded_elements = 1 << rounded_depth;
		printf("rounded elements %d\n ", rounded_elements);
		dim3 fullBlocksPerGrid((rounded_elements + blockSize - 1) / blockSize);
		


		hipMalloc((void**)&dev_buff, 2 * sizeof(float));
		checkCUDAErrorFn("malloc dev_boolbuff in failed!");
		hipMemcpy(dev_buff, in_data, 2 * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAErrorFn("dev_in copy failed!");

		/*hipMalloc((void**)&dev_out, n * sizeof(int));
		checkCUDAErrorFn("malloc dev_out in failed!");
		hipMalloc((void**)&dev_in, rounded_elements * sizeof(int));
		checkCUDAErrorFn("malloc dev_in in failed!");*/


		// is there a way to place this in memory at compile time?
		hipMalloc((void**)&dev_wts, 6 * sizeof(float));
		checkCUDAErrorFn("malloc dev_in in failed!");
		hipMemcpy(dev_wts, weights, 6 * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAErrorFn("dev_weights copy failed!");
		
		// input nodes  
		// each wight has an impact on the other nodes
		for (int i = NUM_LAYERS; i > 0; i--)
		{
			//feed_forward(&in_data[0], &temp[0], (float*)&weights[wt_idx], i, INPUT_NODES);
			//activate_function(&temp[0], &out_data[0], i );
			kernel_feed_forward << < fullBlocksPerGrid, blockSize >> > (i, dev_buff, &dev_wts[wt_idx]);
			checkCUDAErrorFn("feed forward failed!");
			memory_debug_float(i, &dev_buff[0], &host_buff[0]);
			kernel_activate << < fullBlocksPerGrid, blockSize >> > (i, dev_buff);
			checkCUDAErrorFn("activate failed!");
			memory_debug_float(i, dev_buff, host_buff);
			//feed_forward(&out_data[0], &temp[0], (float*)&weights[wt_idx][0], 1,2);
			//activate_function(&temp[0], &out_data[0], 1);
			//std::swap(in_data, out_data);
			wt_idx += 4; // length of array? NUM_NODES* INPUT NODES?
		}

		//error = out_data[0]
        timer().endGpuTimer();
    }
    
	void train_cpu(int n, float *data, float expected)
	{
		
		float weights_layer1[4];
		float weights_layer2[2];

		weights_layer1[0] = 10.1;
		weights_layer1[1] = .9;
		weights_layer1[2] = 20;
		weights_layer1[3] = .87;
		weights_layer2[4] = 41;
		weights_layer2[5] = -54;
		int wt_idx = 0;

		float* hidden_layer = new float[n];
		float* out_put_layer = new float[n];
		float* temp[NUM_LAYERS + 1]; // pointer to arrays
		temp[0] = data;
		temp[1] = hidden_layer;
		temp[2] = out_put_layer;

		float* temp_weights[NUM_LAYERS]; // pointer to arrays
		temp[0] = weights_layer1;
		temp[1] = weights_layer2;


		for (int i = 0; i < NUM_LAYERS; i++)
		{
			feed_forward(temp[i],temp[i+1], temp_weights[i], NUM_LAYERS-i);
		}


		delete hidden_layer;
		delete out_put_layer;
		//back_propagate(in_data,NUM_LAYERS+1,&weights[0],exp,gradient
		

	}
	// TODO: implement required elements for MLP sections 1 and 2 here
}
